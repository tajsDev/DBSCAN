#include "hip/hip_runtime.h"
//example of running the program: ./A5_similarity_search_starter 7490 135000 10000.0 bee_dataset_1D_feature_vectors.txt

#include <stdlib.h>
#include <unistd.h>
#include <stdio.h>
#include <string.h>
#include <omp.h>
#include <iostream>
#include <complex.h>
#include <math.h>
#include <vector>
#include <fstream>
#include <sstream>

#include "disjoint_set.h"

//Error checking GPU calls
#define gpuErrchk(ans) { gpuAssert((ans), __FILE__, __LINE__); }
inline void gpuAssert(hipError_t code, const char *file, int line, bool abort=true)
{
   if (code != hipSuccess)
   {
      fprintf(stderr,"GPUassert: %s %s %d\n", hipGetErrorString(code), file, line);
      if (abort) exit(code);
   }
}

//Mode 1 is the baseline kernel
#define MODE 2

//Define any constants here
//Feel free to change BLOCKSIZE
#define BLOCKSIZE 128


using namespace std;


//function prototypes
//Some of these are for debugging so I did not remove them from the starter file
void warmUpGPU();
void checkParams(unsigned int N, unsigned int DIM);

void importDataset(char * fname, unsigned int N, unsigned int DIM, float * dataset);
void printDataset(unsigned int N, unsigned int DIM, float * dataset);

void computeDistanceMatrixCPU(float * dataset, unsigned int N, unsigned int DIM);
void computeSumOfDistances(float * distanceMatrix, unsigned int N);

void outputDistanceMatrixToFile(float * distanceMatrix, unsigned int N);


//Part 1: Computing the distance matrix 

//Baseline kernel --- one thread per point/feature vector
__global__ void distanceMatrixBaseline(float * dataset, float * distanceMatrix, const unsigned int N, const unsigned int DIM);

//Other kernels that compute the distance matrix (if applicable):



//Part 2: querying the distance matrix
__global__ void queryDistanceMatrixBaseline(float * distanceMatrix, const unsigned int N, const unsigned int DIM, const float epsilon, unsigned int * resultSet);

//Other kernels that query the distance matrix (if applicable):

//DBSCAN PROTOTYPES
__global__ void getNeighbors(float* sortedD, float eps, int DIM, int min_pts, int sortedDim, int *neighborFreqs, int *neighborsArr, int *neighborPos, const int N, const int F);

void expandClusters(int* neighborFreqs, int* neighborsArr, int* neighborPos, int numPoints, int minPts, int* clusterLabels);





int main(int argc, char *argv[])
{
  printf("\nMODE: %d", MODE);
  warmUpGPU(); 



  char inputFname[500];
  unsigned int N=0;
  unsigned int DIM=0;
  float epsilon=0;


  if (argc != 5) {
    fprintf(stderr,"Please provide the following on the command line: N (number of lines in the file), dimensionality (number of coordinates per point), epsilon, dataset filename.\n");
    exit(0);
  }

  sscanf(argv[1],"%d",&N);
  sscanf(argv[2],"%d",&DIM);
  sscanf(argv[3],"%f",&epsilon);
  strcpy(inputFname,argv[4]);

  checkParams(N, DIM);

  printf("\nAllocating the following amount of memory for the dataset: %f GiB", (sizeof(float)*N*DIM)/(1024*1024*1024.0));
  printf("\nAllocating the following amount of memory for the distance matrix: %f GiB", (sizeof(float)*N*N)/(1024*1024*1024.0));
  

  float * dataset=(float*)malloc(sizeof(float*)*N*DIM);
  importDataset(inputFname, N, DIM, dataset);



  //CPU-only mode
  //It only computes the distance matrix but does not query the distance matrix
  if(MODE==0){
    computeDistanceMatrixCPU(dataset, N, DIM);
    printf("\nReturning after computing on the CPU");
    return(0);
  }

  double tstart=omp_get_wtime();

  //Allocate memory for the dataset
  float * dev_dataset;
  gpuErrchk(hipMalloc((float**)&dev_dataset, sizeof(float)*N*DIM));
  gpuErrchk(hipMemcpy(dev_dataset, dataset, sizeof(float)*N*DIM, hipMemcpyHostToDevice));

  //For part 1 that computes the distance matrix
  float * dev_distanceMatrix;
  gpuErrchk(hipMalloc((float**)&dev_distanceMatrix, sizeof(float)*N*N));
  

  //For part 2 for querying the distance matrix
  unsigned int * resultSet = (unsigned int *)calloc(N, sizeof(unsigned int));
  unsigned int * dev_resultSet;
  gpuErrchk(hipMalloc((unsigned int**)&dev_resultSet, sizeof(unsigned int)*N));
  gpuErrchk(hipMemcpy(dev_resultSet, resultSet, sizeof(unsigned int)*N, hipMemcpyHostToDevice));

  
  //Baseline kernels
  if(MODE==1){
  unsigned int BLOCKDIM = BLOCKSIZE; 
  unsigned int NBLOCKS = ceil(N*1.0/BLOCKDIM);
  //Part 1: Compute distance matrix
  distanceMatrixBaseline<<<NBLOCKS, BLOCKDIM>>>(dev_dataset, dev_distanceMatrix, N, DIM);
  //Part 2: Query distance matrix
  queryDistanceMatrixBaseline<<<NBLOCKS,BLOCKDIM>>>(dev_distanceMatrix, N, DIM, epsilon, dev_resultSet);
  }
  
  if (MODE == 2) {
        // Test getNeighbors and expandClusters with the simple CSV
        std::string testFilename = "smiley_face.csv";
        std::vector<float> testData;

        // Read the test CSV file
        std::ifstream testFile(testFilename);
        if (!testFile.is_open()) {
            std::cout << "Failed to open test file: " << testFilename << std::endl;
            return 1;
        }

        std::string line;
        std::getline(testFile, line); // Skip the header line
        while (std::getline(testFile, line)) {
            std::istringstream iss(line);
            std::string value;

            while (std::getline(iss, value, ',')) {
                try {
                    float val = std::stof(value);
                    testData.push_back(val);
                }
                catch (const std::exception& e) {
                    std::cout << "Invalid value: " << value << std::endl;
                }
            }
        }

        testFile.close();

        int numTestPoints = testData.size() / 2;
        float* d_testData;
        int* d_neighborFreqs;
        int* d_neighborsArr;
        int* d_neighborPos;
        hipMalloc(&d_testData, numTestPoints * 2 * sizeof(float));
        hipMalloc(&d_neighborFreqs, numTestPoints * sizeof(int));
        hipMalloc(&d_neighborsArr, numTestPoints * numTestPoints * sizeof(int));
        hipMalloc(&d_neighborPos, numTestPoints * sizeof(int));
        hipMemcpy(d_testData, testData.data(), numTestPoints * 2 * sizeof(float), hipMemcpyHostToDevice);
		
		int* clusterLabels = new int[numTestPoints];

        float testEpsilon = 1.0; // Adjust the epsilon value for the test data
        int testMinPts = 5; // Adjust the minPts value for the test data
        int testDim = 2; // Assuming 2D data for the test CSV
        int testSortedDim = 0; // Sort based on the first dimension (x-coordinate)
		
		int N = numTestPoints;
		int F = numTestPoints* 0.05;

        // Launch the getNeighbors kernel
        getNeighbors<<<(N + BLOCKSIZE - 1) / BLOCKSIZE, BLOCKSIZE>>>(d_testData, testEpsilon, testDim, testMinPts, testSortedDim, d_neighborFreqs, d_neighborsArr, d_neighborPos, N, F);

        // Copy the results back to the host
        int* neighborFreqs = new int[numTestPoints];
        int* neighborsArr = new int[numTestPoints * numTestPoints];
        int* neighborPos = new int[numTestPoints];
        hipMemcpy(neighborFreqs, d_neighborFreqs, numTestPoints * sizeof(int), hipMemcpyDeviceToHost);
        hipMemcpy(neighborsArr, d_neighborsArr, numTestPoints * numTestPoints * sizeof(int), hipMemcpyDeviceToHost);
        hipMemcpy(neighborPos, d_neighborPos, numTestPoints * sizeof(int), hipMemcpyDeviceToHost);

        // Call the expandClusters function on the CPU
        expandClusters(neighborFreqs, neighborsArr, neighborPos, numTestPoints, testMinPts, clusterLabels);

        // Print the cluster labels
        for (int i = 0; i < numTestPoints; i++) {
            std::cout << "Point " << i << " - Cluster: " << clusterLabels[i] << std::endl;
        }

        // Free the allocated memory
        delete[] neighborFreqs;
        delete[] neighborsArr;
        delete[] neighborPos;
		delete[] clusterLabels;
        hipFree(d_testData);
        hipFree(d_neighborFreqs);
        hipFree(d_neighborsArr);
        hipFree(d_neighborPos);
    }

  //Note to reader: you can move querying the distance matrix outside of the mode
  //Part 2: Query distance matrix
  //queryDistanceMatrixBaseline<<<NBLOCKS,BLOCKDIM>>>(dev_distanceMatrix, N, DIM, epsilon, dev_resultSet);
  
  //Copy result set from the GPU
  gpuErrchk(hipMemcpy(resultSet, dev_resultSet, sizeof(unsigned int)*N, hipMemcpyDeviceToHost));

  //Compute the sum of the result set array
  unsigned int totalWithinEpsilon=0;

  //Write code here
  
  printf("\nTotal number of points within epsilon: %u", totalWithinEpsilon);

  double tend=omp_get_wtime();

  printf("\n[MODE: %d, N: %d] Total time: %f", MODE, N, tend-tstart);
  
  

  
  //For outputing the distance matrix for post processing (not needed for assignment --- feel free to remove)
  // float * distanceMatrix = (float*)calloc(N*N, sizeof(float));
  // gpuErrchk(hipMemcpy(distanceMatrix, dev_distanceMatrix, sizeof(float)*N*N, hipMemcpyDeviceToHost));
  // outputDistanceMatrixToFile(distanceMatrix, N);
 

  //Free memory here


  printf("\n\n");
  return 0;
}

//prints the dataset that is stored in one 1-D array
void printDataset(unsigned int N, unsigned int DIM, float * dataset)
{
    for (int i=0; i<N; i++){
      for (int j=0; j<DIM; j++){
        if(j!=(DIM-1)){
          printf("%.0f,", dataset[i*DIM+j]);
        }
        else{
          printf("%.0f\n", dataset[i*DIM+j]);
        }
      }
      
    }  
}




//Import dataset as one 1-D array with N*DIM elements
//N can be made smaller for testing purposes
//DIM must be equal to the data dimensionality of the input dataset
void importDataset(char * fname, unsigned int N, unsigned int DIM, float * dataset)
{
    
    FILE *fp = fopen(fname, "r");

    if (!fp) {
        fprintf(stderr, "Unable to open file\n");
        fprintf(stderr, "Error: dataset was not imported. Returning.");
        exit(0);
    }

    unsigned int bufferSize = DIM*10; 

    char buf[bufferSize];
    unsigned int rowCnt = 0;
    unsigned int colCnt = 0;
    while (fgets(buf, bufferSize, fp) && rowCnt<N) {
        colCnt = 0;

        char *field = strtok(buf, ",");
        double tmp;
        sscanf(field,"%lf",&tmp);
        
        dataset[rowCnt*DIM+colCnt]=tmp;

        
        while (field) {
          colCnt++;
          field = strtok(NULL, ",");
          
          if (field!=NULL)
          {
          double tmp;
          sscanf(field,"%lf",&tmp);
          dataset[rowCnt*DIM+colCnt]=tmp;
          }   

        }
        rowCnt++;
    }

    fclose(fp);

}



void warmUpGPU(){
printf("\nWarming up GPU for time trialing...\n");
hipDeviceSynchronize();
return;
}

/*
sortedD: Dataset sorted along a certain axis
sortedDim: single int that represents which axis is sorted
   - This should be the axis with the biggest range in values
eps: search radius distance
MinPts: minimum number of points in eps to define as part of cluster
DIM: number of dimensions per point

neighborFreqs: a global array which shows how many neighbors each point has 
(size N) 
EX - [2,1,3,4,4...] 
      point 1 has 2 neighbors
	  point 2 has 1 neighbors 
	  point 3 has 3 neighbors ....
	  
neighborsArr: a global array that lists all neighbors, must be used in accordance with neighborFreq
(size N*N*F) where F is ~0.05 which is a percent of how many neighbors a point should have 
EX - [34,26,200,439,23,34590,3459,239,49]
when looking at neighborFreq we can say
	  point 1 has neighbors 34 & 26
	  point 2 has neighbor 200
	  point 3 has neighbors 439, 23, & 34590 ....
	  
neighborPos: a global array which shows the start position of neighborsArr for each point
             This array is populated in getNeighbors, but utilized in expand
(size N)
EX - [0,2,3,6,10]
	  point 1 starts at index 0 of neighborsArr
	  point 2 starts at Index 2 of neighborsArr
	  point 3 starts at index 3 of neighborsArr ....
*/
__global__ void getNeighbors(float* sortedD, float eps, int DIM, int min_pts, int sortedDim, int *neighborFreqs, int *neighborsArr, int *neighborPos, const int N, const int F)
{
	//assign thread ID 0,1,2,3....N
	int tid = threadIdx.x + blockIdx.x * blockDim.x;
	
	//only keep N threads 
	if (tid >= N)
	{
		return;
	}
	
	unsigned int numNeighbors=0;
	float oneDimDistance = 0;
	float fullDistance;
	unsigned int localNeighbors[BLOCKSIZE];
	unsigned int startIndex = 0;
	
	/////////////////////// OBTAINING LOCAL NEIGHBORS  /////////////////////////////
	
	//loop up from threadID element + 1 until difference in sorted dimension values > epsilon
	for (int pointIndex=tid+1; oneDimDistance < eps && pointIndex < N; pointIndex++)            //can optimize by having neighbors >= minpts terminate loop
	{
		float currSumOfDiff = 0;
		
		//this line breaks the loop
		//basically if the difference in the sortedDim of 2 points > eps, no more comparisons needed
		oneDimDistance = sortedD[ tid*DIM+sortedDim ] - sortedD[ pointIndex*DIM+sortedDim ];
		
		//loop through dimensions of points 
		for (int dimIndex = 0; dimIndex < DIM; dimIndex++)
		{
			currSumOfDiff += (sortedD[tid*DIM+dimIndex] - sortedD[pointIndex*DIM+dimIndex]) * 
							  (sortedD[tid*DIM+dimIndex] - sortedD[pointIndex*DIM+dimIndex]);
		}
		fullDistance = sqrt(currSumOfDiff);
		
		if (fullDistance <= eps)
		{
			//neighborArr[tid+neighborIndex] = pointIndex;
			localNeighbors[numNeighbors] = pointIndex;
			numNeighbors++;
		}
	}
	
	float currSumOfDiff = 0;
			
	//loop down from threadID element - 1 until difference in x values > epsilon
	for (int pointIndex=tid-1; oneDimDistance < eps; pointIndex--)
	{
		//this line breaks the loop
		//basically if the difference in the sortedDim of 2 points > eps, no more comparisons needed
		oneDimDistance = sortedD[ tid*DIM+sortedDim ] - sortedD[ pointIndex*DIM+sortedDim ];
		
		//loop through dimensions of points 
		for (int dimIndex = 0; dimIndex < DIM; dimIndex++)
		{
			currSumOfDiff += (sortedD[tid*DIM+dimIndex] - sortedD[pointIndex*DIM+dimIndex]) * 
							  (sortedD[tid*DIM+dimIndex] - sortedD[pointIndex*DIM+dimIndex]);
		}
		fullDistance = sqrt(currSumOfDiff);
		
		if (fullDistance <= eps)
		{
			//neighborArr[tid+neighborIndex] = pointIndex;
			localNeighbors[numNeighbors] = pointIndex;
			numNeighbors++;
		}
	}

	////////////////// POPULATE NEIGHBOR FREQUENCY ARRAY ///////////////////////
	
	//give neighborFreq number of neighbors 
    neighborFreqs[tid] = numNeighbors;
	
	__syncthreads();
	
	//////////////////////// POPULATE NEIGHBORS ARRAY  ////////////////////////
	
	//find starting position
	for (int i = 0; i < tid; i++)
	{
		startIndex += neighborFreqs[i];             //confirm this works
	}
	
	neighborPos[ tid ] = startIndex;
	
	//transfer from registers to global 
	for (int i = startIndex, j=0; i < startIndex + numNeighbors; i++, j++)
	{
	    neighborsArr[ i ] = localNeighbors[j];
	}
	
}



// CPU function for expand clusters using disjoint set
void expandClusters(int* neighborFreqs, int* neighborsArr, int* neighborPos, int numPoints, int minPts, int* clusterLabels)
{
    // Create a disjoint set data structure
    DisjointSet ds(numPoints);

    // Iterate through each point
    for (int i = 0; i < numPoints; i++) {
        if (neighborFreqs[i] >= minPts) {
            // Point forms a cluster
            int startPos = neighborPos[i];
            int endPos = startPos + neighborFreqs[i];

            // Merge the sets containing the point and its neighbors
            for (int j = startPos; j < endPos; j++) {
                int neighbor = neighborsArr[j];
                ds.unionSets(i, neighbor);
            }
        }
    }

    // Assign cluster labels based on the disjoint set
    for (int i = 0; i < numPoints; i++) {
        clusterLabels[i] = ds.findSet(i);
    }
}
